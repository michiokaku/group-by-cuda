#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include"order.h"

__global__ void cuda_order(int *a){
	int tid = threadIdx.x;
	
}

int order(int *a,int length){
	printf("order is running\n");
	int *dev_a;
	hipMalloc((void**)&dev_a, length * sizeof(int)); 
	hipMemcpy(dev_a, a, length * sizeof(int), hipMemcpyHostToDevice);
	return 1;
}