#include<stdio.h>
#include<hip/hip_runtime.h>
#include"order.h"
#define length 256

int * gen(){
	int *a;
	hipHostAlloc( (void**)&a, length* sizeof(int),hipHostMallocDefault );
	for (int i = 0; i < length; ++i)
	{
		a[i] = -1;
	}
	return a ;
}

int main(){
	printf("main is running \n");
	int *a = gen();
	for (int i = 0; i < length; ++i)
	{
		printf("a[%d] = %d \n",i,a[i] );
	}
	order(a,length);
}