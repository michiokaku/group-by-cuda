#include<stdio.h>

struct database{
    int table_lenth;//类别数量
    int num;    //成员数量
    char **table;//类别名称
    char ***data; //数据内容，以字符串存储
    int **data_lenth;//数据长度
};

struct group{
    int **index;//组中元素的索引
    int *lenth;//组的长度
};

void split_line(char **data_str,FILE *fp){
    char buf;
    char ed = -1;
    buf = fgetc(fp);
    while(buf != '\n' && buf != ed){
        int i = 0;
        while(buf != ',' && buf != '\n' && buf != ed){
            i++;
            printf("%c",buf);
            buf = fgetc(fp);
        }
        hipHostAlloc( (void**)&da.table,da.table_lenth * sizeof(char*),hipHostMallocDefault );
        printf("  i=%d ",i);
        if(buf == '\n')break;
        buf = fgetc(fp);
    }
}

database getdata(){
    database da;
    // da.table_lenth = 7 ;
    // da.num = 10;
    // hipHostAlloc( (void**)&da.table,da.table_lenth * sizeof(char*),hipHostMallocDefault );
    FILE *fp = fopen("./data/data.txt","r");
    if (fp == NULL)printf("file open fail !!!!\n");
    char buf;
    buf = fgetc(fp);
    printf("%c",buf);
    char ed = -1;
    da.table_lenth = 1;
    while(buf != '\n'){
        printf("%c",buf);
        if(buf == ',')da.table_lenth++;
        buf = fgetc(fp);
    }
    printf("table_lenth = %d \n",da.table_lenth);
    
    rewind(fp);
    split_line(da.table,fp);
    fclose(fp);
    return da;
}

int main(){
    char key_table[] = "math";//使用这个table来选择分组
    database da = getdata();//函数返回数据库
    // int *index = cuda_sort(da,key_table);//函数返回排序后的索引
    // int group = cuda_group_by(da,key_table);
}

