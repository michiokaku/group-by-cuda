#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include"../add/bitadd.h"
#include"../group_by/group_by.h"

#define block(length) (((length-1)/MAX_THREAD)+1)

void groupfree(group gp){
	hipFree(gp.start);
}

group group_init(int len){
    group gp;
    gp.length = 1;
    hipMalloc((void**)&gp.start,2*sizeof(int));
    int *start_h;
    hipHostAlloc( (void**)&start_h,2 * sizeof(int),hipHostMallocDefault);
    start_h[0] = 0;
    start_h[1] = len;
    hipMemcpy(gp.start,start_h,2*sizeof(unsigned int),hipMemcpyHostToDevice);
    hipHostFree(start_h);
    return gp;
}

__host__ __device__ void ltc(ull l,unsigned char *c,unsigned int p,unsigned char len){
    int a = p*len;
    for(int i=0;i<len;i++){
        c[i+a] = (l>>(8*i))&0xff;
    }
}

__host__ __device__ void ctl(ull &l,unsigned char *c,unsigned int p,unsigned char len){
    int a = p*len;
    l = 0;
    for(int i=0;i<len;i++){
        l += c[i+a]<<(8*i);
    }
}

__host__ __device__ void itc(int in,unsigned char *c,unsigned int p,unsigned char len){
    int a = p*len;
    for(int i=0;i<len;i++){
        c[i+a] = (in>>(8*i))&0xff;
    }
}

__host__ __device__ void cti(int &in,unsigned char *c,unsigned int p,unsigned char len){
    int a = p*len;
    in = 0;
    for(int i=0;i<len;i++){
        in += c[i+a]<<(8*i);
    }
}

__device__ int getppiont(key_s device,int tid){
	int a = device.ppoint_length*tid;
	int out = 0;
	for(int i=0;i<device.ppoint_length;i++){
		out += device.ppoint[i+a]<<(8*i);
	}
	return out;
}

__device__ int getpiont(key_s device,int tid){
    tid = getppiont(device,tid);
    int a = device.point_length*tid;

    int out = 0;
	for(int i=0;i<device.point_length;i++){
		out += device.point[i+a]<<(8*i);
	}

	return out;
}

__device__ int get_next_piont(key_s device,int tid){
	tid = getppiont(device,tid);
	int a = device.point_length*tid;
	a++;

	int out = 0;
	for(int i=0;i<device.point_length;i++){
		out += device.point[i+a]<<(8*i);
	}

	return out;
}

__device__ __host__ int find_group(group gp,int tid){
    int start,end;
    start = 0;
    end = gp.length;
    while((end - start)>1){
        int point = (start+end)/2;
        if(gp.start[point]<=tid)start=point;
        else end = point;
    }
    return start;
}

__device__ unsigned int get_position(bitadd ba,int tid){
    unsigned int position = getba(ba,tid/8);
    int t = tid%8;
    char c = ba.c[tid/8];
    for(int i=0;i<t;i++){
        position += (c>>i)&1;
    }
    return position;
}

__device__ unsigned int get_position_flag(bitadd ba,int tid,unsigned char &flag){
    unsigned int position = getba(ba,tid/8);
    int t = tid%8;
    char c = ba.c[tid/8];
    for(int i=0;i<t;i++){
        position += (c>>i)&1;
    }
    flag = (c>>t)&1;
    return position;
}

__global__ void point_change(group gp,bitadd ba,key_s device,unsigned char *new_pp){
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if(tid<device.point_num){
        unsigned char flag;
        unsigned int position = get_position_flag(ba,tid,flag);
        int local_gp = find_group(gp,tid);
        int start = gp.start[local_gp];
        int next_start = gp.start[local_gp+1];
        unsigned int start_position = get_position(ba,start);
        unsigned int next_position = get_position(ba,next_start);
        int new_position;
        if(flag == 0){
            new_position = tid - position + start_position;
        }
        else{
            new_position = next_start - next_position + position;
        }
        int point;
        cti(point,device.ppoint,tid,device.ppoint_length);
        itc(point,new_pp,new_position,device.ppoint_length);
    }
    if(tid == (gridDim.x*blockDim.x-1)){
    	itc(device.point_num,new_pp,device.point_num,device.ppoint_length);
    }
}

__global__ void empty_check(group gp,bitadd ba,bitadd ba_group,key_s device){//找到下一轮为空的组，空为0，不空为1
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    __shared__ unsigned char add_shared[MAX_THREAD];//末尾两位存两组
    add_shared[threadIdx.x] = 0;
    __syncthreads();
    if(tid<gp.length){
        int len_gp;
        int start = gp.start[tid];
        len_gp = gp.start[tid+1] - start;
        unsigned char flag;
        int position = get_position_flag(ba,start+len_gp,flag)-get_position(ba,start);
        int add0 = 1,add1 = 1;
        if(position == len_gp)add0 = 0;
        if(position == 0)add1 = 0;
        add_shared[threadIdx.x] = add0|(add1<<1);
        __syncthreads();
        if(threadIdx.x<(MAX_THREAD/4)){
            unsigned char c;
            c = add_shared[(threadIdx.x*4)];
            c |= add_shared[(threadIdx.x*4+1)]<<2;
            c |= add_shared[(threadIdx.x*4+2)]<<4;
            c |= add_shared[(threadIdx.x*4+3)]<<6;

            tid = threadIdx.x + ((blockIdx.x*blockDim.x)/4);
            if(tid<ba.length){
                ba_group.c[tid] = c; 
            }   
        }
    }
}

//__global__ void put_new_group(group gp, group newgroup,bitadd ba,bitadd ba_group,key_s device){
//    int tid = threadIdx.x+blockIdx.x*blockDim.x;
//    if(tid < newgroup.length){
//    	unsigned char flag;
//        unsigned int position = get_position_flag(ba_group,tid,flag);
//        if(flag == 1){
//        	int start = gp.start[tid/2];//新的组初始的start的值和原来的组一样
//        	unsigned char branch = tid%2;//判断是新的组是原来组的左半边还是右半边，如果在右边则为一，左边为零
//
//        	if(branch == 1){
//        		int len_gp;
//        		if((tid/2) >= (gp.length-1)){
//        			len_gp = device.point_num-1-start;              //先获取之前组的长度
//        			len_gp -= ba.sum - get_position(ba,tid/2);    //ba.sum - get_position(ba,tid/2)得到原来组中为1项的数量
//        			                                              //再用len_gp减去它得到组中为零项的数量
//        		}
//        		else{
//        			len_gp = gp.start[(tid/2)+1] - start;
//        		    len_gp -= get_position(ba,tid/2+1) - get_position(ba,tid/2);
//        		}
//
//        		start += len_gp;
//        		newgroup.start[position] =  start;
//        	}
//        }
//    }
//}

__global__ void put_new_group(group gp, group newgroup,bitadd ba,bitadd ba_group,key_s device){
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	int half_griddim = gridDim.x/2;
	tid %= half_griddim*blockDim.x;//tid的值是对应的父节点的位置

	if((threadIdx.x == (blockDim.x-1))&&(blockIdx.x == (half_griddim-1))){//左半边最后一个线程给多出的一个组赋值
		newgroup.start[newgroup.length] = gp.start[gp.length];
	}

	if(tid<gp.length){
	//左半边的线程块用于创造左边的子树，右边同理
		unsigned char flag = 0;

		if(blockIdx.x<half_griddim){//左子树
			unsigned int position = get_position_flag(ba_group,tid*2,flag);
			if(flag == 1){//只有flag为1时才继续执行
				newgroup.start[position] = gp.start[tid];//左子树起点不变
			}
		}

		else{//右子树
			unsigned int position = get_position_flag(ba_group,tid*2+1,flag);
			if(flag == 1){
				newgroup.start[position] = gp.start[tid] + (gp.start[tid+1] - gp.start[tid])//父节点的长度
						- (get_position(ba,gp.start[tid+1]) - get_position(ba,gp.start[tid]));//父节点中为1的数量
			}
		}
	}
}

group new_group(group gp,bitadd ba,key_s device){
    bitadd ba_group;
    ba_group.length = ((gp.length-1)/4)+1;
    hipMalloc((void**)&ba_group.c,ba_group.length*MAX_THREAD*sizeof(unsigned char));
    empty_check<<<block(ba_group.length),MAX_THREAD>>>(gp,ba,ba_group,device);
    bit_add(ba_group);
    group newgroup;
    newgroup.length = ba_group.sum;
    hipMalloc((void**)&newgroup.start,(newgroup.length+1)*sizeof(int));//多分配一个方便计算长度。
    put_new_group<<<2*block(gp.length+1),MAX_THREAD>>>(gp,newgroup,ba,ba_group,device);
    bafree(ba_group);
    return newgroup;
}

__global__ void getchanged(key_s device,int *re){
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	if(tid<device.point_num){
		int r;
		cti(r,device.point,tid,device.point_length);
	}
}

void host_point_change(group gp,bitadd ba,key_s &device){
	unsigned char * new_pp;
	hipMalloc((void**)&new_pp,device.ppoint_length*(device.point_num+1)*sizeof(unsigned char));
	point_change<<<block(device.point_num+1),MAX_THREAD>>>(gp,ba,device,new_pp);
	hipFree(device.ppoint);
	device.ppoint = new_pp;
}

group group_by_bitadd(group gp,bitadd ba,key_s &device){
	host_point_change(gp,ba,device);
    group newgroup = new_group(gp,ba,device);
    bafree(ba);
    groupfree(gp);
    return newgroup;
}

//int main(){
//	time_log();
//	group gp = group_init();
//	int len = 1000;
//	unsigned char *c;
//	hipHostAlloc( (void**)&c,len * sizeof(unsigned char),hipHostMallocDefault);
//	for(int i=0;i<len;i++){
//		c[i] = i%256;
//	}
//	bitadd ba;
//	ba.length = len;
//	hipMalloc((void**)&ba.c,len*sizeof(unsigned char));
//	hipMemcpy(ba.c,c,len*sizeof(unsigned char),hipMemcpyHostToDevice);
//	bit_add(ba);
//	key_s device;
//	device.point_num = 7999;
//	time_log();
//	group newgroup = group_by_bitadd(gp,ba,device);
//	int *a;
//	hipHostAlloc( (void**)&a,newgroup.length * sizeof(int),hipHostMallocDefault);
//	hipMemcpy(a,newgroup.start,newgroup.length*sizeof(int),hipMemcpyDeviceToHost);
//	for(int i =0;i<newgroup.length;i++){
//		printf("start[%d] = %d \n",i,a[i]);
//	}
//}

