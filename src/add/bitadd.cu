#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include"bitadd.h"
#include<math.h>

int tlog = 0;
hipEvent_t start;
int n = 664;

int range(){
    n = n*5245 + 12345;
    n = n%32768;
    //printf("n = %d\n",n);
    float a = ((float)n)/32768.0;
    //printf("a = %f\n",a);
    int b = 0;
    if(a>0.5)b=1;
    return b;
}

void gen(unsigned char *c,int len,int length){
    for(int i=0;i<len;i++){
        c[i] = 0;
    }
    for(int i=0;i<length;i++){
        unsigned char cc = range();
        c[(i/8)] |= cc<<(i%8);
    }
    for(int i=0;i<len;i++){
        //printf("c[%d] = %d\n ",i,c[i]);
        //bitprint(c[i]);
    }
}

void bitprint(unsigned char c){
    int a =0;
    for(int i = 0;i<8;i++){
        a = (c>>i)&1;
        printf("%d",a);
    }
    printf("\n");
}

void time_log(){
    if(tlog == 0){
        hipEventCreate(&start);
        hipEventRecord(start,0);
    }
    else{
        float te;
        hipEvent_t end;
        hipEventCreate(&end);
        hipEventRecord(end,0);
        hipEventSynchronize(start);
        hipEventSynchronize(end);
        hipEventElapsedTime(&te,start,end);
        hipEventRecord(start,0);
        printf("time = %f ms\n",te);
    } 
    tlog++;
}

__device__ __host__ unsigned short bts(unsigned char c){
    unsigned short a;
    a = c&1;
    a += (c>>1)&1;
    a += (c>>2)&1;
    a += (c>>3)&1;
    a += (c>>4)&1;
    a += (c>>5)&1;
    a += (c>>6)&1;
    a += (c>>7)&1;
    return a;
}

__device__ int getba(bitadd ba,int index){
    unsigned int a = 0;
    index --;
    if(index < 0)a +=0;
    else a += ba.s[index];

    index = index/(MAX_THREADS_PER_BLOCK*2);
    index--;
    if(index < 0)a += 0;
    else a += ba.i1[index];

    index = index/(MAX_THREADS_PER_BLOCK*2);
    if(index <= 0)a += 0;
    else a += ba.i2[index-1];

    return a;
}

__global__ void dev_get_back(unsigned int *a,bitadd ba){
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if(tid<ba.length){
        a[tid] = getba(ba,tid);
    }
}

__global__ void bit_dev_add(unsigned char * c,unsigned short *sum,int length){
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    __shared__ unsigned short add_shared[MAX_THREADS_PER_BLOCK*2];

    unsigned char rc;
    int tid2 = tid*2;
    if(tid2<length)rc = c[tid2];
    else rc = 0;
    unsigned short ri = bts(rc);
    add_shared[threadIdx.x*2] = ri;
    tid2++;
    if(tid2<length)rc = c[tid2];
    else rc = 0;
    add_shared[(threadIdx.x*2)+1] = ri + bts(rc);
    __syncthreads();

    for(int i = 1;(MAX_THREADS_PER_BLOCK>>i)>0;i++){
        unsigned short ad = ((threadIdx.x<<1)&(0xFFFFFFFF<<(i+1)));
        unsigned short ad2 =0;
        ad |= threadIdx.x&(~(0xFFFFFFFF<<i));
        ad |= 1<<i;
        ad2 = ad&(~(1<<i));
        ad2 |= (~(0xFFFFFFFF<<i));
        add_shared[ad] += add_shared[ad2];
        __syncthreads();
    }
    sum[tid*2] = add_shared[threadIdx.x*2];
    sum[tid*2+1] = add_shared[threadIdx.x*2+1];
}

__global__ void short_dev_add(unsigned short * c,unsigned int *sum,int length){//length是sum的长度
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    __shared__ unsigned int add_shared[MAX_THREADS_PER_BLOCK*2];
    length++;

    int r1 = 0;
    int flag = tid*2 + 1;
    int index;
    if(flag<length){
        index = flag*MAX_THREADS_PER_BLOCK*2;
        index--;
        r1 = c[index];
        add_shared[threadIdx.x*2] = r1;
    }
    else add_shared[threadIdx.x*2] = 0;
    flag++;
    if(flag<length){
        index = flag*MAX_THREADS_PER_BLOCK*2;
        index--;
        r1 += c[index];
        add_shared[threadIdx.x*2+1] = r1;
    }
    else add_shared[threadIdx.x*2+1] = r1;
    
    __syncthreads();
    for(int i = 1;(MAX_THREADS_PER_BLOCK>>i)>0;i++){
        unsigned short ad = ((threadIdx.x<<1)&(0xFFFFFFFF<<(i+1)));
        unsigned short ad2 =0;
        ad |= threadIdx.x&(~(0xFFFFFFFF<<i));
        ad |= 1<<i;
        ad2 = ad&(~(1<<i));
        ad2 |= (~(0xFFFFFFFF<<i));
        add_shared[ad] += add_shared[ad2];
        __syncthreads();
    }
    flag = tid*2;
    sum[tid*2] = add_shared[threadIdx.x*2];
    flag++;
    sum[tid*2+1] = add_shared[threadIdx.x*2+1];
}

__global__ void int_dev_add(unsigned int * c,unsigned int *sum,int length){//到此为止只支持32位的地址寻找
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    __shared__ unsigned int add_shared[MAX_THREADS_PER_BLOCK*2];
    length++;

    int r1 = 0;
    int flag = tid*2 + 1;
    int index;
    if(flag<length){
        index = flag*MAX_THREADS_PER_BLOCK*2;
        index--;
        r1 = c[index];
        add_shared[threadIdx.x*2] = r1;
    }
    else add_shared[threadIdx.x*2] = 0;
    flag++;
    if(flag<length){
        index = flag*MAX_THREADS_PER_BLOCK*2;
        index--;
        r1 += c[index];
        add_shared[threadIdx.x*2+1] = r1;
    }
    else add_shared[threadIdx.x*2+1] = r1;
    
    __syncthreads();
    for(int i = 1;(MAX_THREADS_PER_BLOCK>>i)>0;i++){
        unsigned short ad = ((threadIdx.x<<1)&(0xFFFFFFFF<<(i+1)));
        unsigned short ad2 =0;
        ad |= threadIdx.x&(~(0xFFFFFFFF<<i));
        ad |= 1<<i;
        ad2 = ad&(~(1<<i));
        ad2 |= (~(0xFFFFFFFF<<i));
        add_shared[ad] += add_shared[ad2];
        __syncthreads();
    }
    flag = tid*2;
    sum[tid*2] = add_shared[threadIdx.x*2];
    flag++;
    sum[tid*2+1] = add_shared[threadIdx.x*2+1];
}

void bafree(bitadd &ba){
    hipFree(ba.c);
    hipFree(ba.s);
    hipFree(ba.i1);
    hipFree(ba.i2);
}

void iadd(bitadd &ba){
    printf("in the iadd\n");
    int length = ba.length;
    int block;
    for(int i=0;i<3;i++){
        block = length/(MAX_THREADS_PER_BLOCK*2);
        if((length%(MAX_THREADS_PER_BLOCK*2))>0)block++;
        if(i<2)length = block;
    }
    hipMalloc((void**)&ba.i2,block*MAX_THREADS_PER_BLOCK*2*sizeof(unsigned int));
    hipMemset(ba.i2,0,block*MAX_THREADS_PER_BLOCK*2*sizeof(unsigned int));
    int_dev_add<<<block,MAX_THREADS_PER_BLOCK>>>(ba.i1,ba.i2,length);
}

void sadd(bitadd &ba){
    int length = ba.length;
    int block;
    block = length/(MAX_THREADS_PER_BLOCK*2);
    if((length%(MAX_THREADS_PER_BLOCK*2))>0)block++;
    length = block;
    block = length/(MAX_THREADS_PER_BLOCK*2);
    if((length%(MAX_THREADS_PER_BLOCK*2))>0)block++;
    hipMalloc((void**)&ba.i1,block*MAX_THREADS_PER_BLOCK*2*sizeof(unsigned int));
    hipMemset(ba.i1,0,block*MAX_THREADS_PER_BLOCK*2*sizeof(unsigned int));
    short_dev_add<<<block,MAX_THREADS_PER_BLOCK>>>(ba.s,ba.i1,length);
    if(block>1)iadd(ba);
}

void bit_add(bitadd &ba){
    int block;
    int length = ba.length;
    block = length/(MAX_THREADS_PER_BLOCK*2);
    if((length%(MAX_THREADS_PER_BLOCK*2))>0)block++;
    hipMalloc((void**)&ba.s,block*MAX_THREADS_PER_BLOCK*2*sizeof(unsigned short));
    bit_dev_add<<<block,MAX_THREADS_PER_BLOCK>>>(ba.c,ba.s,length);
    if(block>1)sadd(ba);
    ba.sum = get_sum(ba);
}

void bit_back(bitadd &ba,unsigned int *back){
    int block;
    int length = ba.length;
    block = length/(MAX_THREADS_PER_BLOCK);
    if((length%(MAX_THREADS_PER_BLOCK))>0)block++;

    dev_get_back<<<block,MAX_THREADS_PER_BLOCK>>>(back,ba);
}

unsigned int get_sum(bitadd &ba){
    int length = ba.length;
    int block;

    int t=0;
    for(int i=0;i<3;i++){
        block = length/(MAX_THREADS_PER_BLOCK*2);
        if((length%(MAX_THREADS_PER_BLOCK*2))>0)block++;
        if(i<2)length = block;
        t++;
        if(length<=1)break;
    }
    unsigned int sum = 0;
    int offset = MAX_THREADS_PER_BLOCK*2 -1;
    if(t == 1){
        hipMemcpy(&sum,ba.s+offset,sizeof(unsigned short),hipMemcpyDeviceToHost);
    }
    if(t == 2){
        hipMemcpy(&sum,ba.i1+offset,sizeof(unsigned int),hipMemcpyDeviceToHost);
    }
    if(t == 3){
        hipMemcpy(&sum,ba.i2+offset,sizeof(unsigned int),hipMemcpyDeviceToHost);
    }
    return sum;
}

void ck(int len,unsigned char *o,unsigned int *n){
    int sum = 0;
    int flag = 0;
    for(int i =0;i<len;i++){

        if(sum!=n[i]){
            flag = 1;
            printf("n[%d] = %d\n",i,n[i]);
            printf("o[%d] = %d\n",i,bts(o[i]));
            printf("sum = %d\n",sum);
            printf("has some error in %d \n",i);
        }
        sum+=bts(o[i]);
    }
    printf("check sum = %d\n",sum);
    if(flag == 0)printf("bit add worked succesed\n");
}

// int main(){
//     unsigned int length = 10;
//     float l = log(length)/log(2);
//     printf("log(length) = %f\n",l);
//     unsigned char *c;
//     int len = length/8;
//     if((length%8)>0)len++;
//     printf("len = %d\n",len);
//     hipHostAlloc( (void**)&c,len * sizeof(unsigned char),hipHostMallocDefault);
//     gen(c,len,length);
//     bitadd ba;
//     long long block = 1;
//     block = len/(MAX_THREADS_PER_BLOCK*2);
//     if((len%(MAX_THREADS_PER_BLOCK*2))>0)block++;
//     if(hipSuccess != hipMalloc((void**)&ba.c,len*sizeof(unsigned char))){
//         printf("cudamalloc error\n");
//     }

//     time_log();
//     hipMemset(ba.c,0,len*sizeof(unsigned char));
//     time_log();

//     hipMemcpy(ba.c,c,len*sizeof(unsigned char),hipMemcpyHostToDevice);
//     ba.length = len;
//     time_log();
//     bit_add(ba);
//     printf("bit_add spend :   ");
//     time_log();

//     unsigned int *hb,*db;
//     hipMalloc((void**)&db,(ba.length+1)*sizeof(unsigned int));
//     bit_back(ba,db);
//     hipHostAlloc( (void**)&hb,(ba.length+1) * sizeof(unsigned int),hipHostMallocDefault);
//     hipMemcpy(hb,db,(ba.length+1)*sizeof(unsigned int),hipMemcpyDeviceToHost);
//     ck(len,c,hb);
//     printf("sum = %d\n",ba.sum);
//     bafree(ba);
// }
