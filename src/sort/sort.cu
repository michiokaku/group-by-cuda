#include<stdio.h>
#include<hip/hip_runtime.h>
#include <stdlib.h>

#define length 10
#define length_thread 256

#define test(a){\
	for(int i =0;i<length;i++){\
		printf("a[%d] = %d \n",i,a[i] );\
	}\
}

#define pr_array(a,start,end){\
	for(int i=start;i<=end;i++){\
		printf("a[%d] = %d\n",i,a[i]);\
	}\
}

//return b_end where the vuela a will be put in the new array
//if a equel a vuale which is in the b array,a will in front of the vuale;
#define insert0(a,b,b_start,b_end){\
	while((b_end-b_start)>1){\
		int point = (b_start+b_end)/2;\
		 if(a<=b[point])b_end = point;\
		 else b_start = point;\
	}\
	b_end += (a>b[b_end])-(a<=b[b_start]);\
}\
//if a equel a vuale which is in the b array,a will in back of the vuale;
#define insert1(a,b,b_start,b_end){\
	while((b_end-b_start)>1){\
		int point = (b_start+b_end)/2;\
		 if(a<b[point])b_end = point;\
		 else b_start = point;\
	}\
	b_end += (a>=b[b_end])-(a<b[b_start]);\
}\


int cmp(const void *a,const void *b)
{
    return *(int *)a-*(int *)b;
}

__global__ void pr(int *a){
	int tid = threadIdx.x;
	__shared__ int a_s[length];
	a_s[tid] = a[tid];
	a[tid] = a_s[tid]*2;
}
					
__global__ void merger_thread(int *a,int len){
	__shared__ int a_s[length_thread];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int r1,r2;
	if(tid < len){
		a_s[threadIdx.x] = a[tid];
	}
	r1 = blockDim.x/2;
	int flag = (threadIdx.x>=r1);
	tid = threadIdx.x%r1;
	if((gridDim.x-1) == blockIdx.x)len %= blockDim.x;
	else len = blockDim.x;
	__syncthreads();
	if(threadIdx.x<(len/2)){
		r1 = threadIdx.x*2;
		if(a_s[r1]>a_s[r1+1]){
			r2 = a_s[r1];
			a_s[r1] = a_s[r1+1];
			a_s[r1+1] = r2;
		}
	}
	int loop = 1;
	int x , start , end;
	x = len;
	x = (x/2) + (x%2);
	while(x>1){
		x = (x/2) + (x%2);
		r1 = tid>>loop;
		r1 *= 2;
		start = r1+1-(int)flag;
		r1 += flag;
		end = 1<<loop;
		r1 *= end;
		start *= end;
		r2 = tid % end;
		r1 += r2;
		end += (start-1);
		__syncthreads();
		if(end > len)end = len;
		if(r1 < len){
			r1 = a_s[r1];
			if(flag){
				insert1(r1,a_s,start,end);
			}
			else{
				insert0(r1,a_s,start,end);
			}
			end %= (1<<loop);
			r2 +=end;
			a_s[r2] = r1;
		}
		loop++;
		__syncthreads();
	}
	if(threadIdx.x < len){
		a[threadIdx.x + blockIdx.x * blockDim.x] = a_s[threadIdx.x];
	}
}

int random(int range){
	static int start = 444;
	int d = 233333;
	int k = 33352;
	start = ((start*k)+d)%range;
	return start;
}

int* gen(){
	int *a_h;
	hipHostAlloc( (void**)&a_h, length* sizeof(int),hipHostMallocDefault );
	for (int i = 0; i < length; ++i)
	{
		a_h[i] = random(52);
		printf("a_h[%d] = %d \n",i,a_h[i]);
	}
	return a_h;
}

void sort_int(int *a,int len){
	int block_num = len/length_thread;
	if((len%length_thread)!=0)block_num++;
	merger_thread<<<block_num,length_thread>>>(a,len);
}

void msort(void *a,size_t num,size_t size,int ( * comparator ) ( const void *, const void * ) ){
	
}

int main(){
	/* int *a_h = gen();
	test(a_h);
	qsort(a_h,length,sizeof(int),cmp);
	printf("sorted!!!!!!!\n");
	test(a_h);
	a_h[43]=10;
	a_h[42]=10;
	pr_array(a_h,10,100);
	int start = 10,end = 50;
	insert0(10,a_h,start,end);
	printf("end %d\n",end );
	start = 10;
	end=50;
	insert1(10,a_h,start,end);
	printf("end %d\n",end );
	printf("used ipad pro maked\n");	 */
	int *a_h;
	hipHostAlloc( (void**)&a_h, length* sizeof(int),hipHostMallocDefault );
	for (int i = 0;i<length;i++) {
    	a_h[i] = length - i;
    	printf("a_h[%d] = %d \n",i,a_h[i]);
	}
	int *a_d;
	hipMalloc( (void**)&a_d, length*sizeof(int) );
	hipMemcpy(a_d,a_h, length*sizeof(int),hipMemcpyHostToDevice);
	sort_int(a_d,length);
	hipMemcpy(a_h,a_d,length*sizeof(int),hipMemcpyDeviceToHost);
	for (int i = 0;i<length;i++) {
    	printf("a_h[%d] = %d \n",i,a_h[i]);
	}
}
