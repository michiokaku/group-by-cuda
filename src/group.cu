#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include"add/bitadd.h"
#include"group_by/group_by.h"

#define buflen 100
#define ull unsigned long long int
#define MAX_THREAD 256

void check_group_by_length(group gp,key_s device);
void check_len(unsigned char *len,key_s device);

struct line{
    char **data;//数据内容，以字符串存储
    int *data_lenth;//数据的字符串长度
};

struct database{
    int table_lenth;//类别数量
    ull num;    //成员数量
    line table;//类别名称
    line *ln;//成员的数据
};

int get_line_len(FILE *fp);

int get_block(int length){
    int block = length/MAX_THREAD;
    if((length%MAX_THREAD)!=0)block++;
    return block;
}

unsigned char get_pl(ull str_length){
    unsigned char pl;
    for(int i = 0;i<64;i++){
        if(((str_length>>i)&1)==1){
            pl = i;
        }
    }
    pl++;
    int a=0;
    if((pl%8)>0)a=1;
    pl = (pl/8) +a;
    return pl;
}

void strcopy(char *str1,char *str2){
    int i = 0;
    while(str2[i]&&(i<buflen)){
        str1[i] = str2[i];
        i++;
    }
}

int split_line(char **data_str,int *str_len,FILE *fp,int len){
    if(len != get_line_len(fp)){
        return 0;
    }
    len--;
    char *buf;
    buf = (char *)malloc(buflen*sizeof(char));
    for(int i = 0;i<len;i++){
        if(-1 == fscanf(fp,"%[^,],",buf))printf("wrong in 99");
        int a = strlen(buf);
        a++;
        str_len[i] = a;
        hipHostAlloc( (void**)&data_str[i],a * sizeof(char),hipHostMallocDefault);
        strcopy(data_str[i],buf);
        data_str[i][a-1]=0;
    }
    if(fscanf(fp,"%[^\n]\n",buf)==-1)printf("wrong in 107");
    int a = strlen(buf);
    a++;
    str_len[len] = a;
    hipHostAlloc( (void**)&data_str[len],a * sizeof(char),hipHostMallocDefault);
    strcopy(data_str[len],buf);
    data_str[len][a-1]=0;
    return 1;
}

int get_line_len(FILE *fp){      //获取数据库表头的元素的数量
    long a = ftell(fp); //记录开始位置，结束时要回去
    char buf = fgetc(fp);
    int len= 1;
    if(buf == ',')len++;
    while((buf != -1)&&(buf !='\n')){
        if(buf == ',')len++;
        buf = fgetc(fp);
    }
    fseek(fp,a,0);
    return len;
}

int get_len(FILE *fp){             //获取除了表头以外的行数
    char *buf;
    buf = (char *)malloc(1000*sizeof(char));
    long a = ftell(fp);
    if(fscanf(fp,"%s\n",buf)==-1)printf("wrong in 134");
    int len= -1;             //开始会算上表头，所以要从-1开始
    while(a!=ftell(fp)){
        a = ftell(fp);
        if(fscanf(fp,"%[^\n]\n",buf) == -1);//printf("wrong in 138");
        len++;
    }
    rewind(fp);
    free(buf);
    return len;
}

void goto_next_line(FILE *fp){
    char *buf;
    buf = (char *)malloc(1000*sizeof(char));
    int a = fscanf(fp,"%[^\n]\n",buf);
    if(a==-1)printf("goto_next_line has some problem\n");
    free(buf);
}

database getdata(const char *s){
    database da;
    FILE *fp = fopen(s,"r");
    if (fp == NULL){
        printf("open the \" %s \" failed !!!!\n",s);
        exit(1);
    }
    else printf("open the \" %s \" successful\n",s);
    fgetc(fp);
    da.table_lenth = get_line_len(fp);
    da.num = get_len(fp);  
    hipHostAlloc( (void**)&da.table.data,da.table_lenth * sizeof(char*),hipHostMallocDefault);
    hipHostAlloc( (void**)&da.table.data_lenth,da.table_lenth * sizeof(int),hipHostMallocDefault);
    if(0 == split_line(da.table.data,da.table.data_lenth,fp,da.table_lenth)){
        printf("The table has some proble!!!!\n");
        goto_next_line(fp);
    }

    hipHostAlloc( (void**)&da.ln,da.num * sizeof(line),hipHostMallocDefault);
    for(int i = 0;i<da.num;i++){
        hipHostAlloc( (void**)&da.ln[i].data,da.table_lenth * sizeof(char*),hipHostMallocDefault);
        hipHostAlloc( (void**)&da.ln[i].data_lenth,da.table_lenth * sizeof(int),hipHostMallocDefault);
        if(0 == split_line(da.ln[i].data,da.ln[i].data_lenth,fp,da.table_lenth)){
            printf("The number of field in line<%d>is not same with the table!!!!\n",i);
            goto_next_line(fp);
        }
    }
    fclose(fp);
    return da;
}

ull keylen(line *host,int len,int k){
    ull kl=0;
    for(int i=0;i<len;i++){
        kl+=(host[i].data_lenth[k]-1);
    }
    return kl;
}

int keyscp(line *host,key_s &khost,ull kl,int len,int k){
    int p = 0;
    khost.point_num = (len+1);//多给一个位置，用下一项的位置减当前项的位置来获取长度，最后一个没有下一项，需要多给一个
    khost.str_length = kl;
    khost.point_length = get_pl(kl);
    hipHostAlloc( (void**)&khost.point,khost.point_num * khost.point_length * sizeof(char),hipHostMallocDefault);
    khost.ppoint_length = get_pl(khost.point_num);
    hipHostAlloc( (void**)&khost.ppoint,khost.point_num*khost.ppoint_length * sizeof(char),hipHostMallocDefault);
    for(int i=0;i<len;i++){
        ltc(p,khost.point,i,khost.point_length);
        ltc(i,khost.ppoint,i,khost.ppoint_length);
        for(int j=0;j<(host[i].data_lenth[k]-1);j++){
            if(p>=kl)return 0;
            khost.key_str[p] = host[i].data[k][j];
            p++;
        }
    }
    ltc(kl,khost.point,len,khost.point_length);
    ltc(len,khost.ppoint,len,khost.ppoint_length);
    return 1;
}

void key_htd(line *host,key_s &device,int len,int k){
    ull kl = keylen(host,len,k);
    key_s khost;
    hipHostAlloc( (void**)&khost.key_str,kl * sizeof(char),hipHostMallocDefault);
    keyscp(host,khost,kl,len,k);
    device = khost;
    hipMalloc( (void**)&device.point,device.point_num*device.point_length*sizeof(unsigned char));
    hipMalloc( (void**)&device.ppoint,device.point_num*device.ppoint_length*sizeof(unsigned char));
    hipMalloc( (void**)&device.key_str,device.str_length*sizeof(char));

    hipMemcpy(device.point,khost.point, device.point_num*device.point_length*sizeof(unsigned char),hipMemcpyHostToDevice);
    hipMemcpy(device.ppoint,khost.ppoint, device.point_num*device.ppoint_length*sizeof(unsigned char),hipMemcpyHostToDevice);
    hipMemcpy(device.key_str,khost.key_str, device.str_length*sizeof(char),hipMemcpyHostToDevice);

    hipHostFree(khost.point);
    hipHostFree(khost.ppoint);
    hipHostFree(khost.key_str);
}

void printline(line ln,int len){
    for(int i = 0;i<len;i++){
        printf("%s  ",ln.data[i]);
    }
    printf("\n");
}

void printdata(database da){
    printline(da.table,da.table_lenth);
    for(int i=0;i<da.num;i++){
        printline(da.ln[i],da.table_lenth);
    }
}

int getkey(database da,char *key_table){
    int k = -1;
    for(int i=0;i<da.table_lenth;i++){
        if(strcmp(da.table.data[i],key_table)==0){
            k = i;
            break;
        }
    }
    if(k == -1)printf("can not found the \" %s \"\n",key_table);
    else printf("found the key \" %s \", the index is %d\n",key_table,k);
    return k;
}

__global__ void dev_getlen(key_s device,unsigned char *len){
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    if(tid<(device.point_num)){
        int len1,len2;
        cti(len1,device.point,tid,device.point_length);
        cti(len2,device.point,(tid+1),device.point_length);
        len[tid] = len2 - len1;
    }
}

unsigned char * host_getlen(key_s device){
    unsigned char *len;
    hipMalloc( (void**)&len,(device.point_num)*sizeof(unsigned char));
    int block = device.point_num/MAX_THREAD;
    if((device.point_num%MAX_THREAD)!=0)block++;
    dev_getlen<<<block,MAX_THREAD>>>(device,len);
    return len;
}

__global__ void gbl_bit(key_s device,unsigned char *len,int b,int length,bitadd ba){
    int tid = threadIdx.x+blockIdx.x*blockDim.x;
    __shared__ unsigned char add_shared[MAX_THREAD];
    add_shared[threadIdx.x] = 0;
    if(tid<length){
        add_shared[threadIdx.x] = (len[getppiont(device,tid)]>>b)&1;
        __syncthreads();
        unsigned char r;
        if(threadIdx.x <(MAX_THREAD/8)){
            r = add_shared[(threadIdx.x*8)];
            r |= add_shared[(threadIdx.x*8)+1]<<1;
            r |= add_shared[(threadIdx.x*8)+2]<<2;
            r |= add_shared[(threadIdx.x*8)+3]<<3;
            r |= add_shared[(threadIdx.x*8)+4]<<4;
            r |= add_shared[(threadIdx.x*8)+5]<<5;
            r |= add_shared[(threadIdx.x*8)+6]<<6;
            r |= add_shared[(threadIdx.x*8)+7]<<7;

            tid = threadIdx.x + ((blockIdx.x*blockDim.x)/8);
            if(tid<ba.length){
                ba.c[tid] = r; 
            }   
        }
    }
}

group group_by_length(unsigned char *len,key_s &device){//len是字符串的长度
    group gp = group_init(device.point_num);
    int length = device.point_num;
    for(int i=0;i<8;i++){   //char有八个比特，所以循环八次
        bitadd ba;
        ba.length = ((length-1)/8)+1;
        if((length%8)!=0)ba.length++;
        hipMalloc( (void**)&ba.c,ba.length*sizeof(unsigned char));
        gbl_bit<<<get_block(length),MAX_THREAD>>>(device,len,7-i,length,ba);      //把第7-i位的值取出来放入ba.s中
        bit_add(ba);                       //使用bit_add函数计算出结果
        gp = group_by_bitadd(gp,ba,device);     //根据bit_add的结果分组
//        check_group_by_length(gp,device);
    }
    return gp;
}

group group_by(database da,char *key_table){
    int k = getkey(da,key_table);
    key_s device;
    key_htd(da.ln,device,da.num,k);
    device.point_num--;//长度多分配了一个，多出的一个给最后一个计算长度用，为了方便point_num不包括多分配的一个，所以--
    unsigned char *len = host_getlen(device);//首先先要获取每段字符串的长度，根据长度先分组
    check_len(len,device);
    group gp = group_by_length(len,device);//返回一个根据长度分组的结果
    check_group_by_length(gp,device);
    return gp;
}

void getpath(char * arg,char * c){
    int len = strlen(arg);
    len --;
    for(int i=0;i<=len;i++){
        if(arg[len-i] == '/'){
            arg[len-i+1] = 0;
            break;
        }
    }
    strcpy(c,arg);
    strcat(c,"../data/data.txt");
}

void key_s_dth(key_s device,key_s &host){
	host = device;
	hipHostAlloc( (void**)&host.point,(host.point_length+1) * host.point_num * sizeof(unsigned char),hipHostMallocDefault);
	hipHostAlloc( (void**)&host.ppoint,(host.ppoint_length+1) * host.point_num * sizeof(unsigned char),hipHostMallocDefault);
	hipHostAlloc( (void**)&host.key_str,(host.str_length+1) * sizeof(unsigned char),hipHostMallocDefault);

	hipMemcpy(host.point,device.point, (host.point_num+1)*host.point_length*sizeof(unsigned char),hipMemcpyDeviceToHost);
	hipMemcpy(host.ppoint,device.ppoint, (host.point_num+1)*host.ppoint_length*sizeof(unsigned char),hipMemcpyDeviceToHost);
	hipMemcpy(host.key_str,device.key_str, (host.str_length+1)*sizeof(char),hipMemcpyDeviceToHost);
}

int check_get_length(key_s h,int tid){
	int pp;
	cti(pp,h.ppoint,tid,h.ppoint_length);
	int p1,p2;
	cti(p1,h.point,pp,h.point_length);
	cti(p2,h.point,pp+1,h.point_length);
	int p = p2-p1;
	return p;
}

void check_len(unsigned char *len,key_s device){
	unsigned char *h_len;
	hipHostAlloc( (void**)&h_len,device.point_num * sizeof(unsigned char),hipHostMallocDefault);
	hipMemcpy(h_len,len, device.point_num*sizeof(unsigned char),hipMemcpyDeviceToHost);
	key_s host;
	key_s_dth(device,host);

	int flag = 0;
	for(int i=0;i<(host.point_num-1);i++){
		unsigned char clen = check_get_length(host,i);
		if(clen != h_len[i]){
			printf("h_len[%d] = %d\n",i,h_len[i]);
			flag = 1;
		}
	}
	if(flag == 0)printf("len is right\n");
}

void check_group_by_length(group gp,key_s device){
	unsigned char *h_len;
	key_s host;
	key_s_dth(device,host);
	hipHostAlloc( (void**)&h_len,host.point_num * sizeof(unsigned char),hipHostMallocDefault);

	int last = -1;
	for(int i=0;i<(host.point_num);i++){
		h_len[i] = check_get_length(host,i);
		if(last != h_len[i]){
			last = h_len[i];
			printf("h_len[%d] = %d\n",i,h_len[i]);
		}
	}
}

int main(int argc, char * argv[]){
    char key_table[] = "python";//使用这个table来选择分组
    char c[100];
    getpath(argv[0],c);//工作目录与执行文件的相对路径可以根据argv[0]得到
    database da = getdata(c);//函数返回数据库
    group gp;
    if(argc>1)gp = group_by(da,argv[1]);
    else gp = group_by(da,key_table);
    int *a;
    hipHostAlloc( (void**)&a,gp.length * sizeof(int),hipHostMallocDefault);
    hipMemcpy(a,gp.start,gp.length*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<gp.length;i++){
    	printf("a[%d] = %d \n",i,a[i]);
    }
}

