#include<stdio.h>
#include<hip/hip_runtime.h>

#define length 256

__global__ void pr(int *a){
	int tid = threadIdx.x;
	__shared__ int a_s[length];
	a_s[tid] = a[tid];
	a[tid] = a_s[tid]*2;
}

int random(int range){
	static int star = 444;
	int d = 233333;
	int k = 33352;
	star = ((star*k)+d)%range;
	return star;
}

int* gen(){
	int *a_h;
	hipHostAlloc( (void**)&a_h, length* sizeof(int),hipHostMallocDefault );
	for (int i = 0; i < length; ++i)
	{
		a_h[i] = random(100);
		printf("a_h[%d] = %d \n",i,a_h[i]);
	}
	return a_h;
}

int main(){
	int *a_h = gen();
	int *a_d;
	hipMalloc((void**) &a_d, sizeof(int) * length);
	hipMemcpy(a_d, a_h, sizeof(int) * length,hipMemcpyHostToDevice); 
	pr<<<1,length>>>(a_d);
	hipMemcpy(a_h, a_d, sizeof(int) * length,hipMemcpyDeviceToHost);
	for (int i = 0; i < length; ++i)
	{
		printf("a_h[%d] = %d \n",i,a_h[i]);
	}
}