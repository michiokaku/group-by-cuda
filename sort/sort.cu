#include<stdio.h>
#include<hip/hip_runtime.h>

#define length 256

__global__ void pr(int *a){
	int tid = threadIdx.x;
	__shared__ int a_s[length];
	a_s[tid] = a[tid];
	a[tid] = a_s[tid]*2;
}

int random(int range){
	static int star = 444;
	int d = 233333;
	int k = 33352;
	star = ((star*k)+d)%range;
	return star;
}

int* gen(){
	int *a_h;
	hipHostAlloc( (void**)&a_h, length* sizeof(int),hipHostMallocDefault );
	for (int i = 0; i < length; ++i)
	{
		a_h[i] = random(100);
		printf("a_h[%d] = %d \n",i,a_h[i]);
	}
	return a_h;
}

int main(){
	


}